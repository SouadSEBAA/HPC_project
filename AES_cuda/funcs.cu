#include "hip/hip_runtime.h"
#include "funcs.h"

#define THREAD_PER_BLOCK 512
#define NB_BLOCKS 1

__global__ void device_encrypt (unsigned char *plaintext , unsigned char *ciphertext , unsigned char *nonce, unsigned char *expansion, size_t N)
{
    int i = threadIdx.x;
    unsigned char* block = (unsigned char*) malloc(BLOCK_SIZE * sizeof(unsigned char));
    unsigned char* counter = (unsigned char*) malloc(BLOCK_SIZE * sizeof(unsigned char));
    memcpy(counter, nonce, BLOCK_SIZE);

    if (i < N && i < THREAD_PER_BLOCK)
    {
        get_ith_item(block, plaintext, i, BLOCK_SIZE);
        if (DEBUG) 
        {
            printf("Block before encryption/decryption i = %d : ",i); 
            print_data(block, BLOCK_SIZE);
        }

        increment_counter(counter, i);
        if (DEBUG) 
        {
            printf("Counter : ");
            print_data(counter, BLOCK_SIZE); 
        }

        encrypt_block(block, expansion, counter);
        if (DEBUG) 
        {
            printf("Encrypted/Decryption block : ");
            print_data(block, BLOCK_SIZE); 
        }

        memcpy(ciphertext + BLOCK_SIZE*i, block, BLOCK_SIZE);

    }

    free(block);
    block = NULL;
    free(counter);
    counter = NULL;

}

void kernel_encrypt_wrapper(unsigned char *plaintext , unsigned char *ciphertext , unsigned char *nonce, unsigned char *expansion, size_t N)
{
    unsigned char *gpu_expansion , *gpu_ciphertext , *gpu_plaintext, *gpu_nonce;
    hipEvent_t start, stop;
    float elapsedTime;

    //allocate memory in the device
    int size_data = N * BLOCK_SIZE * sizeof(unsigned char);
    int size_expansion = 4*(ROUNDS_NUMBER+1)*4 * sizeof(unsigned char);
    hipMalloc ((void**) &gpu_ciphertext, size_data);
    hipMalloc ((void**) &gpu_plaintext, size_data);
    hipMalloc ((void**) &gpu_nonce, BLOCK_SIZE);
    hipMalloc ((void**) &gpu_expansion, size_expansion);

    //copy data 
    hipMemcpy(gpu_plaintext, plaintext, size_data, hipMemcpyHostToDevice);
    hipMemcpy(gpu_nonce, nonce, BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpu_expansion, expansion, size_expansion, hipMemcpyHostToDevice);

    //for time
    hipEventCreate(&start);
    hipEventRecord(start,0);
    hipEventCreate(&stop);

    //execute on device
    device_encrypt <<<NB_BLOCKS,THREAD_PER_BLOCK>>> (gpu_plaintext, gpu_ciphertext, gpu_nonce, gpu_expansion, N);

    //copy results
    hipMemcpy(ciphertext, gpu_ciphertext, size_data, hipMemcpyDeviceToHost);

    //calculate time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);

    printf("time = %f ms\n" ,elapsedTime);

    hipFree(gpu_ciphertext);
    hipFree(gpu_plaintext);
    hipFree(gpu_nonce);
    hipFree(gpu_expansion);
}


unsigned char *encrypt_cuda(unsigned char *plaintext, size_t size, unsigned char *key, unsigned char *counter, bool debug) {
    size_t nblocks;
    unsigned char *expansion, *ciphertext;

    expansion = key_expansion(key, BLOCK_SIZE);
    if (DEBUG) {
        printf("Expanded key : ");
        print_data(expansion, BLOCK_SIZE*ROUNDS_NUMBER + 1);
        printf("Nonce : ");
        print_data(counter, BLOCK_SIZE); 
    }

    nblocks = size / BLOCK_SIZE + (size % BLOCK_SIZE != 0);

    ciphertext = (unsigned char*) malloc(nblocks * BLOCK_SIZE * sizeof(unsigned char));
   
    kernel_encrypt_wrapper(plaintext, ciphertext, counter, expansion, nblocks);

    return ciphertext;
}

unsigned char *decrypt_cuda(unsigned char *ciphertext, size_t size, unsigned char *key, unsigned char *counter, bool debug) {
    return encrypt_cuda(ciphertext, size, key, counter, debug);
}