#include "hip/hip_runtime.h"
#include "funcs.h"

#define THREAD_PER_BLOCK 512
#define NB_BLOCKS 65535

__global__ void device_encrypt (unsigned char *plaintext , unsigned char *ciphertext , unsigned char *nonce, unsigned char *expansion, size_t N)
{
    int i = blockIdx.x * blockDim.x + blockIdx.y + threadIdx.x;
    unsigned char* block = (unsigned char*) malloc(BLOCK_SIZE * sizeof(unsigned char));
    unsigned char* counter = (unsigned char*) malloc(BLOCK_SIZE * sizeof(unsigned char));
    memcpy(counter, nonce, BLOCK_SIZE);

    if (i < N && i < THREAD_PER_BLOCK * NB_BLOCKS)
    {
        get_ith_item(block, plaintext, i, BLOCK_SIZE);
        if (DEBUG) 
        {
            printf("Block before encryption/decryption i = %d : ",i); 
            print_data(block, BLOCK_SIZE);
        }

        increment_counter(counter, i);
        if (DEBUG) 
        {
            printf("Counter : ");
            print_data(counter, BLOCK_SIZE); 
        }

        encrypt_block(block, expansion, counter);
        if (DEBUG) 
        {
            printf("Encrypted/Decryption block : ");
            print_data(block, BLOCK_SIZE); 
        }

        memcpy(ciphertext + BLOCK_SIZE*i, block, BLOCK_SIZE);

    }

    free(block);
    block = NULL;
    free(counter);
    counter = NULL;

}

void kernel_encrypt_wrapper(unsigned char *plaintext , unsigned char *ciphertext , unsigned char *nonce, unsigned char *expansion, size_t N)
{
    unsigned char *gpu_expansion , *gpu_ciphertext , *gpu_plaintext, *gpu_nonce;

    //allocate memory in the device
    int size_data = N * BLOCK_SIZE * sizeof(unsigned char);
    int size_expansion = 4*(ROUNDS_NUMBER+1)*4 * sizeof(unsigned char);
    hipMalloc ((void**) &gpu_ciphertext, size_data);
    hipMalloc ((void**) &gpu_plaintext, size_data);
    hipMalloc ((void**) &gpu_nonce, BLOCK_SIZE);
    hipMalloc ((void**) &gpu_expansion, size_expansion);

    //copy data 
    hipMemcpy(gpu_plaintext, plaintext, size_data, hipMemcpyHostToDevice);
    hipMemcpy(gpu_nonce, nonce, BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpu_expansion, expansion, size_expansion, hipMemcpyHostToDevice);

    //execute on device
    int blocks = N / THREAD_PER_BLOCK + (N % THREAD_PER_BLOCK != 0);
    device_encrypt <<<blocks,THREAD_PER_BLOCK>>> (gpu_plaintext, gpu_ciphertext, gpu_nonce, gpu_expansion, N);

    //copy results
    hipMemcpy(ciphertext, gpu_ciphertext, size_data, hipMemcpyDeviceToHost);

    hipFree(gpu_ciphertext);
    hipFree(gpu_plaintext);
    hipFree(gpu_nonce);
    hipFree(gpu_expansion);
}


unsigned char *encrypt_cuda(unsigned char *plaintext, size_t size, unsigned char *key, unsigned char *counter, bool debug) {
    size_t nblocks;
    unsigned char *expansion, *ciphertext;

    expansion = key_expansion(key, BLOCK_SIZE);
    if (DEBUG) {
        printf("Expanded key : ");
        print_data(expansion, BLOCK_SIZE*ROUNDS_NUMBER + 1);
        printf("Nonce : ");
        print_data(counter, BLOCK_SIZE); 
    }

    nblocks = size / BLOCK_SIZE + (size % BLOCK_SIZE != 0);

    ciphertext = (unsigned char*) malloc(nblocks * BLOCK_SIZE * sizeof(unsigned char));
   
    kernel_encrypt_wrapper(plaintext, ciphertext, counter, expansion, nblocks);

    return ciphertext;
}

unsigned char *decrypt_cuda(unsigned char *ciphertext, size_t size, unsigned char *key, unsigned char *counter, bool debug) {
    return encrypt_cuda(ciphertext, size, key, counter, debug);
}